#include "hip/hip_runtime.h"
/*
 * spmv_driver.cu
 * Copyright (C) 2018 
 * 	P Sadayappan (saday) <psaday@gmail.com>
 * 	Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>

void check_vec(double* a, double *b, unsigned int n, bool quit_on_err = true ) {
    for (unsigned int i = 0; i < n; ++i) {
        if(std::abs(a[i] - b[i]) > 1e-1) {
            std::cerr << "Possible error at " << i << std::endl;
            if(quit_on_err)
                exit(-1);
        }
    }
}

void init_vec(double *a, unsigned int n, double offset) {
    for (unsigned int i = 0; i < n; ++i) {
        a[i]  = i + offset;
    }
}

void host_spmv(CSR mat, double* vec_in, double *vec_out  ) {
    for (unsigned int r = 0; r < mat.nrows; ++r) {
        unsigned int row_start = mat.row_indx[r];
        unsigned int row_end = mat.row_indx[r + 1];
        vec_out[r] = 0;

        for (unsigned int j = row_start; j < row_end; ++j) {

            unsigned int col_id = mat.col_id[j];
            double val = mat.values[j];
            vec_out[r] += val * vec_in[col_id];
        }

    }
}

int main(int argc, char *argv[]) {
    if(argc < 2) {
        std::cerr << "Missing filename" << std::endl;
        exit(-1);
    }

    CSR mat = read_matrix_market_to_CSR(argv[1]);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << std::endl;

    double *vec_in = (double*)malloc(mat.ncols * sizeof(double));
    double *vec_out = (double*)malloc(mat.nrows * sizeof(double));

    init_vec(vec_in, mat.ncols, 1.0);

    host_spmv(mat, vec_in, vec_out);


    std::cout << "replace one argument to the below function with the values from gpu " << std::endl;
    check_vec(vec_out, vec_out, mat.nrows);


    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);
    return 0;
}
